#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cstddef>

#include "common.h"
#include "block_reduce.h"
#include "kernels.h"

namespace br { namespace cuda {

__global__ void nll_loss(const int* ids, const float* logsoftmax, float *output, float *dout, int n, int vocab , float loss_scale) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    /*
    float value = 0.0;
    if ( i < n ) {
        int index = ids[i];
        value = logsoftmax[ i * vocab +  index]; 
    }

    float reduce_val[1] = {value};
    blockReduce<ReduceType::kSum, 1>(reduce_val);
    
    if ( i == 0 ) {
        *output = reduce_val[0];
    }
    */
    
    float value = 0.0;
    if ( i < n ) {
        int index = ids[i];
        if ( index >= 0 && index < vocab ) {
            value = logsoftmax[ i * vocab +  index]; 
            dout[i * vocab + index] = -1.0 * loss_scale;
        }
    }
    
    float reduce_val[1] = {value};
    lightseq::cuda::blockReduce<lightseq::cuda::ReduceType::kSum, 1>(reduce_val);
    
    if ( i == 0 ) {
        *output = reduce_val[0];
    }
}

int nllloss_forward(const int* ids, const float* logsoftmax, float *output, float *dout, int n, int vocab, float loss_scale, hipStream_t stream) {
    dim3 block_size(256);
	dim3 num_of_blocks((n + block_size.x - 1) / block_size.x);

    nll_loss <<< num_of_blocks, block_size, 0, stream >>> (ids, logsoftmax, output, dout, n, vocab, loss_scale);
 
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(-1);
    }
 
    return 0;
}

}}
