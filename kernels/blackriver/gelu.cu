#include "hip/hip_runtime.h"
#include "kernels.h"

/*
def bloom_gelu_forward(x: torch.Tensor) -> torch.Tensor:
    """
    Custom bias GELU function. Adapted from Megatron-DeepSpeed code. Here we use a simple implementation (inference) to
    make the model jitable.

    Args:
        x (`torch.tensor`, *required*):
            input hidden states
    """
    return x * 0.5 * (1.0 + torch.tanh(0.79788456 * x * (1 + 0.044715 * x * x)))


def bloom_gelu_back(g: torch.Tensor, x: torch.Tensor) -> torch.Tensor:
    """
    gradient of tanh approximation of gelu gradient of actual gelu is: 0.5 * (1. + torch.erf(x * 0.70710678)) +
    0.3989423 * x * torch.exp(-0.5 * x * x)

    Args:
        g (`torch.tensor`, *required*):
            gradient output tensor
        x (`torch.tensor`, *required*):
            input tensor
    """
    x = x[0]  # x is a tuple of 1 element, needs to unpack it first
    tanh_out = torch.tanh(0.79788456 * x * (1 + 0.044715 * x * x))
    # sqrt(2/pi) * 3 * 0.044715 -> 0.1070322243
    ff = 0.5 * x * ((1 - tanh_out * tanh_out) * (0.79788456 + 0.1070322243 * x * x)) + 0.5 * (1 + tanh_out)
    return ff * g
*/

namespace br { namespace cuda {

__global__ void gelu(float* target, const float* src, int nElementNumber) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nElementNumber) {
        float value = src[i];
        target[i] = value * (0.5F + 0.5F * tanhf(value * (0.79788456F + 0.03567741F * value * value)));
    }
}

int gelu_forward(const float* src, float* target, int nElementNumber, hipStream_t stream) {
    dim3 block_size(256);
	dim3 num_of_blocks((nElementNumber + block_size.x - 1) / block_size.x);

    gelu <<< num_of_blocks, block_size, 0, stream >>> (target, src, nElementNumber);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(-1);
    }
    
    return 0;
}

__global__ void gelu_bw(const float* out_g, const float* xi, float* x_g, int nElementNumber) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nElementNumber) {
        float x = xi[i];
        float tanh_out = tanh(0.79788456 * x * (1 + 0.044715 * x * x));
        float ff = 0.5 * x * ((1 - tanh_out * tanh_out) * (0.79788456 + 0.1070322243 * x * x)) + 0.5 * (1 + tanh_out);
        x_g[i] = ff * out_g[i];
    }
}

int gelu_backward(const float* out_g, const float* xi, float* x_g, int nElementNumber, hipStream_t stream) {
    dim3 block_size(256);
	dim3 num_of_blocks((nElementNumber + block_size.x - 1) / block_size.x);

    gelu_bw <<< num_of_blocks, block_size, 0, stream >>> (out_g, xi, x_g, nElementNumber);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(-1);
    }
    
    return 0;
}

}}
